#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> 

int * myloadFile(int n,char *s){
	int i;
	int *a=(int *)malloc(sizeof(int)*n);
	for(i=0;i<n;i++)
	{
		a[i]=i;
	}
	return a;
	
} 
void display(int *a,int n){
	int i;
	for(i=0;i<n;i++)
	{
		printf("%d %d \n",i,a[i]);
 	}
		 


}


hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	int low=2031*i;
	int high=2031*(i+1)-1;	
	for(int i=low;i<=high;i++){
		c[i] = a[i] + b[i];
	}

	printf("%d %d ",low,high);
}

int main()
{
    int arraySize =2079744;
    const int *a=myloadFile(arraySize,"input1.txt");
    const int *b=myloadFile(arraySize,"input2.txt");
    int *c=myloadFile(arraySize,"input2.txt");
	int i;
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
	for(int i=0;i<arraySize;i++){
		printf("%d %d %d\n",a[i],b[i],c[i]);
   	} 
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 1024>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
 
    return cudaStatus;
}
